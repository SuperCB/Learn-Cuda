
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <hip/hip_runtime.h>

#include <hipblas.h>

using namespace std;
#define IDX2C(i, j, ld) (((i) * (ld)) + (j))
#define TILE_SIZE 16

void printPlainMatrix(const float *matrix, const int H, const int W) {
    for (int i = 0; i < H; i++) {
        for (int j = 0; j < W; j++) {
            std::cout << std::fixed << std::setw(8) << std::setprecision(4) << matrix[IDX2C(i, j, W)];
        }
        std::cout << std::endl;
    }
}


__global__ void operator_matmul_h(const float *input1, const float *input2,
                                  float *output, int height, int k, int width) {
    __shared__ float shared_input1[TILE_SIZE][TILE_SIZE];
    __shared__ float shared_input2[TILE_SIZE][TILE_SIZE];


    int bx = blockIdx.y;
    int by = blockIdx.x;
    int tx = threadIdx.y;
    int ty = threadIdx.x;

    int row = bx * TILE_SIZE + tx;
    int col = by * TILE_SIZE + ty;
    float v = 0;

    for (int i = 0; i < (int) (ceil((float) k / TILE_SIZE)); i++) {
        if (i * TILE_SIZE + ty < k && row < height)
            shared_input1[tx][ty] = input1[row * k + i * TILE_SIZE + ty];
        else
            shared_input1[tx][ty] = 0;

        if (i * TILE_SIZE + tx < k && col < width)
            shared_input2[tx][ty] = input2[(i * TILE_SIZE + tx) * width + col];
        else
            shared_input2[tx][ty] = 0;
        __syncthreads();

        for (int j = 0; j < TILE_SIZE; j++)
            v += shared_input1[tx][j] * shared_input2[j][ty];
        __syncthreads();
    }

    if (row < height && col < width) output[row * width + col] = v;
}


int main() {


    // Prepare input matrices
    float *A, *B, *C;
    int M, N, K;
    float alpha, beta;

    M = 20;
    N = 20;
    K = 30;
    alpha = 1.f;
    beta = 0.f;


    hipMallocManaged((void **) &A, sizeof(float) * M * K);
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < K; j++) {
            A[IDX2C(i, j, K)] = i + 1;
        }
    }

    /*
      A:
      1.0000  1.0000  1.0000
      2.0000  2.0000  2.0000
    */


    hipMallocManaged((void **) &B, sizeof(float) * K * N);
    for (int i = 0; i < K; i++) {
        for (int j = 0; j < N; j++) {
            B[IDX2C(i, j, N)] = i + 1;
        }
    }

    hipMallocManaged((void **) &C, sizeof(float) * M * N);
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            C[IDX2C(i, j, N)] = 1;
        }
    }

    dim3 dim_block(TILE_SIZE, TILE_SIZE);
    dim3 dim_grid(ceil((float) M / TILE_SIZE), ceil((float) N / TILE_SIZE));
    operator_matmul_h<<<dim_grid, dim_block>>>(A, B, C,
                                               N, K, M);


    hipblasHandle_t handle;

    // Prepare input matrices


    // create cuBLAS handle
    if (hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "CUBLAS initialization failed" << std::endl;
        return EXIT_FAILURE;
    }


    float *c1;
    hipMallocManaged((void **) &c1, sizeof(float) * M * N);
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            c1[IDX2C(i, j, N)] = 1;
        }
    }
    hipblasSgemm(handle,
                HIPBLAS_OP_T,// 通过转置变为逻辑上的matrix
                HIPBLAS_OP_T, // 同上
                M, // 逻辑上A和C的row的个数
                N, // 逻辑上B和C的column的个数
                K, // 逻辑上A的column个数和B的row 的个数
                &alpha,
                A,
                K, //由于使用了转置，这里填写A的列数
                B,
                N, // 由于使用了转置, 这里填写B的列数
                &beta,
                c1,
                N);

    hipDeviceSynchronize();
    printPlainMatrix(C, M, N);


    printPlainMatrix(c1, M, N);

    // 从 显存 中取出运算结果至 内存中去
//    cublasGetVector (
//            M*M,    //  要取出元素的个数
//            sizeof(float),    // 每个元素大小
//            c1,    // GPU 端起始地址
//            1,    // 连续元素之间的存储间隔
//            h_C,    // 主机端起始地址
//            1    // 连续元素之间的存储间隔
//    );

    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; j++) {
            if (C[i * N + j] != c1[j * M + i])
                cout << "fuck" << endl;
        }
    }
    cout << endl;


    hipFree(A);
    hipFree(B);
    hipFree(C);
    hipFree(c1);

    hipblasDestroy(handle);

    return 0;
}